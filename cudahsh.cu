#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "stringgen.h"
#include "cudahsh.h"
#include "md5.h"

static struct timeval timer() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return tp;
}

static double toSeconds(struct timeval tp) {
   return ((double) (tp.tv_sec) + 1e-6 * tp.tv_usec);
}

char* removeNewline(char* str) {
   if(str != NULL && !strcmp(str + (strlen(str) - 1), "\n")) {
      str[(strlen(str) - 1)] = 0;
   }
   
   return str;
}

int main(int argc, char** argv) {
    if(argc < 3) {
        printf("Usage: cudahsh.out <input file> <output file>\n");
        printf("Use a CUDA compatible GPU to crack the hashes in the input file and output the keys to the output file.\n");
        printf("Uses brute force and only works on keys made from letters and numbers.\n");
        return 1;
    }

    FILE* input = fopen(argv[1], "r");
    if(!input) {
    	printf("Error opening input file %s!\n", argv[1]);
    	return 1;
    }

    FILE* output = fopen(argv[2], "w");
    if(!output) {
    	printf("Error opening output file %s!\n", argv[2]);
    	fclose(input);
    	return 1;
    }
    
    double elt = 0.0;
    struct timeval start;
    start = timer();
	
    char* key = (char*) calloc(KEY_LENGTH + 2, sizeof(char));
    char* hash = (char*) calloc(HASH_LENGTH + 1, sizeof(char));
    char* keyArr = (char*) calloc(N * (KEY_LENGTH + 2), sizeof(char));
    char* gpuKey;
    hipMalloc(&gpuKey, (KEY_LENGTH + 2) * sizeof(char));
	char* gpuHash;
    hipMalloc(&gpuHash, (HASH_LENGTH + 1) * sizeof(char));
    char* gpuKeyArr;
    hipMalloc(&gpuKeyArr, N * (KEY_LENGTH + 2) * sizeof(char));
    bool stopAll;
    bool* gpuStopAll;
    hipMalloc(&gpuStopAll, sizeof(bool));
    
    int blocksPerGrid = ceil(N / 512.0);
    int threadsPerBlock = N / blocksPerGrid;
    //printf("%d\n", threadsPerBlock);

    for(int i; fgets(hash, HASH_LENGTH, input); i++) {
    	removeNewline(hash);

    	//brute force the hash from buf and print the result to a line in output
    	charlist_t* sequence;
	    sequence = new_charlist_element();

	    stopAll = false;
	    hipMemset(gpuStopAll, false, sizeof(bool));

	    while(!stopAll && strlen(key) <= KEY_LENGTH)
	    {
	    	//generate an array of keys to hash
	    	for(int curKey = 0; curKey < N && strlen(key) <= KEY_LENGTH; curKey++)
		    {
		    	//printf("%d\n", curKey);
		    	sprint_charlist(key, sequence);
		    	memcpy(&keyArr[curKey * (KEY_LENGTH + 2)], md5_pad(key), (KEY_LENGTH + 2) * sizeof(char));
		    	//printf("%s %d\n", &keyArr[curKey * (KEY_LENGTH + 2)], curKey);
		    	
		        next(sequence);
		    }
		    
		    hipMemcpy(gpuKeyArr, keyArr, N * (KEY_LENGTH + 2) * sizeof(char), hipMemcpyHostToDevice);
   		    hipMemcpy(gpuHash, hash, N * (KEY_LENGTH + 2) * sizeof(char), hipMemcpyHostToDevice);

		    //hash the keys and check the hashes
	    	//getHash<<<blocksPerGrid, threadsPerBlock>>>(hash, key);
			md5_calculate(blocksPerGrid, threadsPerBlock, gpuHash, gpuKeyArr, gpuKey, gpuStopAll);
	    	
	    	hipMemcpy(&stopAll, gpuStopAll, sizeof(bool), hipMemcpyDeviceToHost);
	    }

		hipMemcpy(key, gpuKey, (KEY_LENGTH + 2) * sizeof(char), hipMemcpyDeviceToHost);
		memcpy(key, md5_unpad(key), KEY_LENGTH + 2);
		key[KEY_LENGTH] = 0;
		free_charlist(sequence);
		
		struct timeval end = timer();
        elt = toSeconds(end) - toSeconds(start);
		
		fprintf(output, "%s\n", key);
		printf("Time taken: %3.3lf s.\n", elt);

	    
    }
    
    fclose(input);
    fclose(output);
    free(key);
    free(hash);
    hipFree(gpuKeyArr);
    return 0;
}
