#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "stringgen.h"

#define HASH_LENGTH 64 //number of characters, not counting null (extra space is allocated for null)
#define KEY_LENGTH 4 //number of characters, not counting null (extra space is allocated for null)

typedef enum {false, true} bool;

char* removeNewline(char* str) {
   if(str != NULL && !strcmp(str + (strlen(str) - 1), "\n")) {
      str[(strlen(str) - 1)] = 0;
   }
   
   return str;
}

//used to hash a test key - the vast majority of the hashes won't match the one we're looking for.
__global__ void getHash(char* hash, const char* src) {
    //get the hash using whichever algorithm we choose
    memcpy(hash, src, HASH_LENGTH + 1);
}

int main(int argc, char** argv) {
    if(argc < 3) {
        printf("Usage: cudahsh.out <input file> <output file>\n");
        printf("Use a CUDA compatible GPU to crack the hashes in the input file and output the keys to the output file.\n");
        printf("Uses brute force and only works on keys made from letters and numbers.\n");
        return 1;
    }

    FILE* input = fopen(argv[1], "r");
    if(!input) {
    	printf("Error opening input file %s!\n", argv[1]);
    	return 1;
    }

    FILE* output = fopen(argv[2], "w");
    if(!output) {
    	printf("Error opening output file %s!\n", argv[2]);
    	fclose(input);
    	return 1;
    }

    char* buf = (char*) calloc(HASH_LENGTH + 1, sizeof(char));
    char* key = (char*) calloc(KEY_LENGTH + 2, sizeof(char));
    char* hash = (char*) calloc(HASH_LENGTH + 1, sizeof(char));

    for(int i; fgets(buf, HASH_LENGTH, input); i++) {
    	removeNewline(buf);

    	//brute force the hash from buf and print the result to a line in output
    	charlist_t* sequence;
	    sequence = new_charlist_element();

	    bool stopAll = false;

	    while(!stopAll && strlen(key) <= KEY_LENGTH)
	    {
	    	//generate an array of keys to hash
	    	while(strlen(key) <= KEY_LENGTH && )
		    {
		    	sprint_charlist(key, sequence);
		    	//printf("%s\n", key);
		    	getHash(hash, key);
		    	if(!strcmp(hash, buf)) {
		    		printf("%s\n", hash);
		    		stopAll = true;
		    	}

		        next(sequence);
		    }

		    //hash the keys and check the hashes
	    	sprint_charlist(key, sequence);
	    	//printf("%s\n", key);
	    	getHash<<1, 1>>(hash, key);
	    	if(!strcmp(hash, buf)) {
	    		printf("%s\n", hash);
	    		stopAll = true;
	    	}

	        next(sequence);
	    }

	    free_charlist(sequence);
    }
    
    fclose(input);
    fclose(output);
    free(buf);
    free(key);
    free(hash);
    return 0;
}
